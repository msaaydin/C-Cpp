#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "kernel.cuh"


__global__ void cuda_hello() {
	int id = threadIdx.x;
	printf("Hello World from GPU! blockid.x = %d threadidx = %d\n",blockIdx.x,id);
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
__global__ void increment_by_one_kernel(int* vals) {
	vals[threadIdx.x] += 1;
}

void callHello() {
	cuda_hello <<<2, 10 >>>();
	hipDeviceSynchronize();
}

void increment_by_one(int* a) {
	int* a_d;
	hipMalloc(&a_d, 1);
	hipMemcpy(a_d, a, 1, hipMemcpyHostToDevice);
	increment_by_one_kernel << <1, 1 >> >(a_d);
	hipMemcpy(a, a_d, 1, hipMemcpyDeviceToHost);
	hipFree(a_d);
}